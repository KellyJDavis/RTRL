//
//  arguments.cpp
//  RTLRL
//
//  Created by Kelly Davis on 23/05/2015.
//
//


#include <exception>

#include <cusp/coo_matrix.h>
#include <cusp/io/matrix_market.h>

#include "arguments.h"


namespace rtrl
{
    Arguments::Arguments(int argc,char *argv[])
    :   m_message(),
        m_is_valid(true),
        m_d(),
        m_x(),
        m_w(),
        m_y_1() 
    {
        // Check if the number of arguments is incorrect
        if(argc != 5)
        {
            // Set message
            m_message = "Incorrect number of arguments.";

            // Flag validity
            m_is_valid = false;

            // Return
            return;
        }

        // Try to load matrices
        try
        {
            cusp::io::read_matrix_market_file(m_d, argv[1]);
            cusp::io::read_matrix_market_file(m_x, argv[2]);
            cusp::io::read_matrix_market_file(m_w, argv[3]);
            cusp::io::read_matrix_market_file(m_y_1, argv[4]);
        }
        catch(const std::exception& exception)
        {
            // Set message
            m_message = exception.what();

            // Flag validity
            m_is_valid = false;
        }
    }
}
