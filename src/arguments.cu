//
//  arguments.cpp
//  RTLRL
//
//  Created by Kelly Davis on 23/05/2015.
//
//


#include <sstream>

#include <cusp/io/matrix_market.h>

#include "arguments.h"


namespace rtrl
{
    // Utility functions to check arguments
    bool is_argument_number_correct(int argc, const char *prog, std::string &out_message);
    bool is_t_dim_correct(size_t d_num_rows, size_t x_num_rows, std::string &out_message);
    bool is_n_dim_correct(size_t d_num_cols, size_t w_num_rows, std::string &out_message);
    bool are_n_and_m_dim_correct(size_t d_num_cols, size_t x_num_cols, size_t w_num_cols, std::string &out_message);

    Arguments::Arguments(int argc,char *argv[])
    :   m_message(),
        m_is_valid(true),
        m_d(),
        m_x(),
        m_w()
    {
        // Check if the number of arguments is incorrect
        if(!(m_is_valid = is_argument_number_correct(argc,argv[0],m_message)))
          return;

        // Load matrices
        read_matrix_market_file(m_d, argv[1]);
        read_matrix_market_file(m_x, argv[2]);
        read_matrix_market_file(m_w, argv[3]);

        // Check T dimension
        if(!(m_is_valid = is_t_dim_correct(m_d.num_rows,m_x.num_rows,m_message)))
          return;

        // Check n dimension
        if(!(m_is_valid = is_n_dim_correct(m_d.num_cols,m_w.num_rows,m_message)))
          return;

        // Check n and m dimensions
        if(!(m_is_valid = are_n_and_m_dim_correct(m_d.num_cols,m_x.num_cols,m_w.num_cols,m_message)))
          return;
    }

    // Utiltiy function to check argument number
    bool is_argument_number_correct(int argc, const char *prog, std::string &out_message)
    {
        // Check if the number of arguments is incorrect
        if(argc != 4)
        {
            // Create string stream
            std::stringstream message;

            // Set message
            message << "The proper calling syntax is:\n\n";
            message << " '" << prog << " <target values> <external inputs> <initial weight matrix>'\n\n";
            message << "where all passed parameters are matrix market files.";

            // Convert message to string
            out_message = message.str();

            // Return
            return false;
        }

        // Return
        return true;
    }

    // Utiltiy function to check T dimension 
    bool is_t_dim_correct(size_t d_num_rows, size_t x_num_rows, std::string &out_message)
    {
        // Check T dimension
        if(d_num_rows != x_num_rows)
        {
            // Create string stream
            std::stringstream message;

            // Set message
            message << "The row count " << d_num_rows << " of <target values> ";
            message << "is not equal to the row count " << x_num_rows << " of <external input>.";

            // Convert message to string
            out_message = message.str();

            // Return
            return false;
        }

        // Return
        return true;
    }

    // Utiltiy function to check n dimension 
    bool is_n_dim_correct(size_t d_num_cols, size_t w_num_rows, std::string &out_message)
    {
        // Check n dimension
        if(d_num_cols != w_num_rows)
        {
            // Create string stream
            std::stringstream message;

            // Set message
            message << "The column count " << d_num_cols << " of <target values> ";
            message << "is not equal to the row count " << w_num_rows << " of <initial weight matrix>.";

            // Convert message to string
            out_message = message.str();

            // Return
            return false;
        }

        // Return
        return true;
    }

    // Utiltiy function to check n and m dimensions 
    bool are_n_and_m_dim_correct(size_t d_num_cols, size_t x_num_cols, size_t w_num_cols, std::string &out_message)
    {
        // Check n and m dimensions
        if((d_num_cols + x_num_cols) != w_num_cols)
        {
            // Create string stream
            std::stringstream message;

            // Set message
            message << "The column count " << d_num_cols << " of <target values> plus ";
            message << "the column count " << x_num_cols << " of <external input> ";
            message << "is not equal to the column count " << w_num_cols << " of <initial weight matrix>.";

            // Convert message to string
            out_message = message.str();

            // Return
            return false;
        }

        // Return
        return true;
    }
}
