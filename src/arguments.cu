//
//  arguments.cpp
//  RTLRL
//
//  Created by Kelly Davis on 23/05/2015.
//
//


#include <exception>

#include <cusp/coo_matrix.h>
#include <cusp/io/matrix_market.h>

#include "globals.h"
#include "arguments.h"


namespace rtrl
{
    // Define globals
    cusp::coo_matrix<uint_fast64_t, float, cusp::host_memory> g_d;
    cusp::coo_matrix<uint_fast64_t, float, cusp::host_memory> g_x;
    cusp::coo_matrix<uint_fast64_t, float, cusp::device_memory> g_w;
    cusp::coo_matrix<uint_fast64_t, float, cusp::device_memory> g_y_1;

    Arguments::Arguments(int argc,char *argv[])
    :   m_message(),
        m_is_valid(true)
    {
        // Check if the number of arguments is incorrect
        if(argc != 5)
        {
            // Set message
            m_message = "Incorrect number of arguments.";

            // Flag validity
            m_is_valid = false;

            // Return
            return;
        }

        // Try to load matrices
        try
        {
            cusp::io::read_matrix_market_file(g_d, argv[1]);
            cusp::io::read_matrix_market_file(g_x, argv[2]);
            cusp::io::read_matrix_market_file(g_w, argv[3]);
            cusp::io::read_matrix_market_file(g_y_1, argv[4]);
        }
        catch(const std::exception& exception)
        {
            // Set message
            m_message = exception.what();

            // Flag validity
            m_is_valid = false;
        }
    }
}
